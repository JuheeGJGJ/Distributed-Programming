#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MATRIX_SIZE 4096
#define BLOCK_SIZE 32

__global__ void matrixMultiplication(float* A, float* B, float* C) {
    extern __shared__ float shared_A[BLOCK_SIZE][BLOCK_SIZE];
    extern __shared__ float shared_B[BLOCK_SIZE][BLOCK_SIZE];
    extern __shared__ float shared_C[BLOCK_SIZE][BLOCK_SIZE];

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    shared_C[threadIdx.y][threadIdx.x] = 0;

    if (row < MATRIX_SIZE && col < MATRIX_SIZE) {
        for (int i = 0; i < MATRIX_SIZE / BLOCK_SIZE; i++) {
            shared_A[threadIdx.y][threadIdx.x] = A[row * MATRIX_SIZE + threadIdx.x + i * BLOCK_SIZE];
            shared_B[threadIdx.y][threadIdx.x] = B[(threadIdx.y + i * BLOCK_SIZE) * MATRIX_SIZE + col];
            __syncthreads();

            for (int j = 0; j < BLOCK_SIZE ; j += 2) {
                shared_C[threadIdx.y][threadIdx.x] += shared_A[threadIdx.y][j] * shared_B[j][threadIdx.x];
                shared_C[threadIdx.y][threadIdx.x] += shared_A[threadIdx.y][j + 1] * shared_B[j + 1][threadIdx.x];
            }
        }
    }

    C[row * MATRIX_SIZE + col] = shared_C[threadIdx.y][threadIdx.x];
}

void matrixMultiplication_CPU(float* matrix_A, float* matrix_B, float* matrix_CPU, float* matrix_C) {
    int i, j, k;

    for (i = 0; i < MATRIX_SIZE; i++) {
        for (j = 0; j < MATRIX_SIZE; j++) {
            matrix_CPU[i * MATRIX_SIZE + j] = 0;

            for (k = 0; k < MATRIX_SIZE; k++) {
                matrix_CPU[i * MATRIX_SIZE + j] += matrix_A[i * MATRIX_SIZE + k] * matrix_B[k * MATRIX_SIZE + j];
            }

            if (matrix_C[i * MATRIX_SIZE + j] != matrix_CPU[i * MATRIX_SIZE + j]) {
                printf("C : %f CPU : %f\n", matrix_C[i * MATRIX_SIZE + j], matrix_CPU[i * MATRIX_SIZE + j]);
                printf("wrong calculation!\n");

                return;
            }
        }
        printf("line %d correct\n", i);
    }
    printf("No error!\n");
}

int main()
{
    srand((unsigned int)time(NULL));

    /* 4096 * 4096 matrix + randomly generated floating number */
    float* matrix_A, * matrix_B, * matrix_C, * matrix_CPU;
    matrix_A = (float*)malloc(sizeof(float) * MATRIX_SIZE * MATRIX_SIZE);
    matrix_B = (float*)malloc(sizeof(float) * MATRIX_SIZE * MATRIX_SIZE);
    matrix_C = (float*)malloc(sizeof(float) * MATRIX_SIZE * MATRIX_SIZE);
    matrix_CPU = (float*)malloc(sizeof(float) * MATRIX_SIZE * MATRIX_SIZE);

    int i;
    for (i = 0; i < MATRIX_SIZE * MATRIX_SIZE; i++) {
        //matrix_A[i] = (float)rand()/((float)RAND_MAX/10);
        //matrix_B[i] = (float)rand()/((float)RAND_MAX/10);
        matrix_A[i] = (i % 7);
        matrix_B[i] = (i % 11);
    }

    /* set device memory */
    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE);
    hipMemcpy(d_A, matrix_A, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice);
    hipMalloc(&d_B, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE);
    hipMemcpy(d_B, matrix_B, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice);
    hipMalloc(&d_C, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE);

    /* start timer */
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    /* matrix multiplication */
    dim3 dimGrid(MATRIX_SIZE / BLOCK_SIZE, MATRIX_SIZE / BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    matrixMultiplication << <dimGrid, dimBlock >> > (d_A, d_B, d_C);

    /* end timer */
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time :  %f ms\n", time);

    /* copy to host */
    hipMemcpy(matrix_C, d_C, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyDeviceToHost);

    /* check multiplication result (debugging) */
    //matrixMultiplication_CPU(matrix_A, matrix_B, matrix_CPU, matrix_C);

    /* free memory */
    free(matrix_A);
    free(matrix_B);
    free(matrix_C);

    return 0;
}